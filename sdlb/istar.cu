#include <hip/hip_runtime.h>
				// Stops underlining of __global__
	// Stops underlining of threadIdx etc.

#include <stdio.h>

__global__ void compute_istar_cuda(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}