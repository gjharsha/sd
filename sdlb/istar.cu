#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>
#define DBL_MAX         1.7976931348623158e+308 /* max value */
extern "C" {
#include "cuda_wrapper.h"
}



__global__ void compute_istar_cuda(sigma_type *sigma, delta_type *delta, i_type *istar, double *argmax, vector pi_Tbar_x, vector Xvect, int rv_cols, int ictr, int count)
{

	int sig_pi=0, del_pi=0, c=0;
	double largest_temp;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < count)
	{
		double largestSoFar = -DBL_MAX;
		for (sig_pi = 0; sig_pi < sigma->cnt; sig_pi++)
		{
			if (sigma->ck[sig_pi] <= ictr)
			{
				/* Find the row in delta corresponding to this row in sigma */
				del_pi = sigma->lamb[sig_pi];
				/* Start with (Pi x Rbar) + (Pi x Romega) + (Pi x Tbar) x X */
				largest_temp = sigma->val[sig_pi].R + delta->val[del_pi][idx].R - pi_Tbar_x[sig_pi];

				/* Subtract (Pi x Tomega) x X. Multiply only non-zero VxT values */
				for (c = 1; c <= rv_cols; c++)
					largest_temp -= delta->val[del_pi][idx].T[c] * Xvect[delta->col[c]];

				if (largest_temp > largestSoFar)
				{
					largestSoFar = largest_temp;
					istar[idx].sigma = sig_pi;
					istar[idx].delta = del_pi;
					argmax[idx] = largestSoFar;
				}

			}
		}
	}
}

extern "C"
void launch_kernel(sigma_type *sigma, delta_type *delta, i_type *istar, double *argmax, vector pi_Tbar_x, vector Xvect, int rv_cols, int ictr, int count)
{
	compute_istar_cuda<<<(count / 640) + 1, 640>>>(sigma, delta, istar, argmax, pi_Tbar_x, Xvect, rv_cols, ictr, count);
	hipDeviceSynchronize();
}

__global__ void foo()
{
}

extern "C" 
void CudaMain(void)
{
	foo<<<1, 1>>>();
}
